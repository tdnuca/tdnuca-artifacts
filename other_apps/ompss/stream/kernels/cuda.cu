
#include <hip/hip_runtime.h>
extern "C" {
__global__ void init_kernel (double *a, double *b, double *c, int bs, int j)
{
   unsigned int i = blockIdx.x * blockDim.x + threadIdx.x + j;
  if ( i >= j+bs ) return;
  a[i] = 1.0;
  b[i] = 2.0;
  c[i] = 0.0;
  a[i] = 2.0E0 * a[i];
}

__global__ void copy_kernel (double *a, double *c, int bs, int j)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x + j;
  if ( i >= j+bs ) return;
  c[i] = a[i];
}

__global__ void scale_kernel (double *b, double *c, double scalar, int bs, int j)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x + j;
  if ( i >= j+bs ) return;
  b[i] = scalar * c[i];
}

__global__ void add_kernel (double *a, double *b, double *c, int bs, int j)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x + j;
  if ( i >= j+bs ) return;
  c[i] = a[i] + b[i];
}
__global__ void triad_kernel (double *a, double *b, double *c, double scalar, int bs, int j)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x + j;
  if ( i >= j+bs ) return;
  a[i] = b[i] + scalar * c[i];
}
}
